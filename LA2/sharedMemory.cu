
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include<math.h>

__global__ void computePrefixSum(double* arr, double *prefixArraySums, int maxElementsInArray, int N){
    
    __shared__ int Arr[320000];
    for(int i=0; i<320000; i++){
        Arr[i] = arr[i];
    }
    
    
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        // indexTillEnd = N-1; //check later
        indexTillEnd = N;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    
    double sum = 0;
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        sum += Arr[i];
        arr[i] = sum;
    }
    prefixArraySums[threadIndex] = sum;
}

__global__ void computeFinalSum(double* arr, double *prefixArraySums, int maxElementsInArray, int N){
    int threadIndex = threadIdx.x;
    int indexToStartFrom = threadIndex * maxElementsInArray;
    int indexTillEnd;
    if((threadIndex+1)*maxElementsInArray -1 >= N){
        indexTillEnd = N;
    }
    else{
        indexTillEnd = (threadIndex+1)*maxElementsInArray -1;
    }
    double numberToAdd = prefixArraySums[threadIndex];
    for(int i=indexToStartFrom; i<=indexTillEnd; i++){
        arr[i] += numberToAdd;
    }
}

int main()
{
    char line[100] = {0};
    int N = 320000;

    // divide into 32 sub-arrays
    int totalNumberOfSubArrays = 32;
    double doubleType;
    
    size_t size = N * sizeof(double);
    size_t size2 = totalNumberOfSubArrays * sizeof(double);
    
    double *arr;
    hipMallocManaged(&arr, size);
    
    double *prefixArraySums;
    hipMallocManaged(&prefixArraySums, size2);
    int maxElementsInArray = N/totalNumberOfSubArrays;
    if(N%totalNumberOfSubArrays>0){
        maxElementsInArray++;
    }
    
    for(int i=0; i<N; i++){
        arr[i] = 1;
    }
    
    computePrefixSum<<<1, totalNumberOfSubArrays>>>(arr, prefixArraySums, maxElementsInArray, N);
    
    hipDeviceSynchronize();
    
    // prefix sum
    double prefSumArray = 0;
    for(int j=0; j<totalNumberOfSubArrays; j++){
        double temp = prefSumArray;
        prefSumArray += prefixArraySums[j];
        prefixArraySums[j] = temp;
    }
    
    computeFinalSum<<<1, totalNumberOfSubArrays>>>(arr, prefixArraySums, maxElementsInArray, N);
    hipDeviceSynchronize();
    hipFree(arr);  
    hipFree(prefixArraySums);
}