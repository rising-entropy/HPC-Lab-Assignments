
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void computeConvolutionValue(int mRows, int mCols, int mWidth, int convRows, int convCols, int *m, int *c, int *output, int outputCols, int outputRows){
    int rowIndex=threadIdx.x, colIndex=blockIdx.x;
    // we need the index of m[rowIndex][colIndex][0]
    int res = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                //m[rowIndex+k][colIndex+j][i]*c[k][j][i]
                int indexOfC = i*(convCols*convRows) + (j*convRows) + k;
                int indexOfM = i*(mRows*mCols) + ((colIndex+j)*mRows) + (rowIndex+k);
                res += (c[indexOfC] * m[indexOfM]);
            }
        }
    }
    
    int indexToFillInValue = rowIndex + (outputRows*colIndex);
    output[indexToFillInValue] = res;
}

int main(){
    char line[100] = {0};
    // 2D Conv width is 1
    int mRows=512, mCols=512, mWidth=1, convRows=3, convCols=3;
    int outputRows, outputCols;
    int i=0;
    
//     int m[mRows][mCols][mWidth], c[convRows][convCols][mWidth];
    int *m, *c, *output;
    hipMallocManaged(&m, sizeof(int)*mRows*mCols*mWidth);
    hipMallocManaged(&c, sizeof(int)*convRows*convCols*mWidth);

    outputRows = mRows - convRows + 1;
    outputCols = mCols - convCols + 1;
//     int output[outputRows][outputCols];
    hipMallocManaged(&output, sizeof(int)*outputRows*outputCols);
    
    int m_input[mRows][mCols][mWidth], c_input[convRows][convCols][mWidth];
    
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                m_input[k][j][i] = 1;
            }
        }
    }
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                c_input[k][j][i] = 2;
            }
        }
    }
    int val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<convCols; j++){
            for(int k=0; k<convRows; k++){
                c[val] = c_input[k][j][i];
                val++;
            }
        }
    }
    
    val = 0;
    for(int i=0; i<mWidth; i++){
        for(int j=0; j<mCols; j++){
            for(int k=0; k<mRows; k++){
                m[val] = m_input[k][j][i];
                val++;
            }
        }
    }
    
    
    int m2[mRows][mCols][mWidth], c2[convRows][convCols][mWidth];
    
    for(int i=0; i<mWidth*convCols*convRows; i++){
        
        int z = i/(convCols*convRows);
        int v = i - z*(convCols*convRows);
        int y = v/convRows;
        v -= y*convRows;
        int x = v;
        c2[x][y][z] = c[i];
    }
    
    for(int i=0; i<mWidth*mCols*mRows; i++){
        
        int z = i/(mCols*mRows);
        int v = i - z*(mCols*mRows);
        int y = v/mRows;
        v -= y*mRows;
        int x = v;
        m2[x][y][z] = m[i];
    }
    
    computeConvolutionValue<<<outputCols, outputRows>>>(mRows, mCols, mWidth, convRows, convCols, m, c, output, outputCols, outputRows);
    hipDeviceSynchronize();
    
    hipFree(m);
    hipFree(c);
    hipFree(output);
    
    return 0;
}
